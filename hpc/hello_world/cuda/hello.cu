
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void cuda_hello(){
    // wont be printed
    printf("Hello World from GPU!\n");
}

int main() {
    // 1 block , 1 thread
    cuda_hello<<<1,1>>>(); 
    printf("Hello from cpu\n");
    return 0;
}