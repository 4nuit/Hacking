#include <stdlib.h>
#include <stdio.h>
#include <time.h>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>
#include <hiprand/hiprand_kernel.h>


#define BLOCK_SIZE 516
#define NB_BLOCK 258
#define N 10 //N*NB_BLOCK*BLOCK_SIZE ~ 1.07e6

__global__ void cuda_inCircle(double *d_in_circle, hiprandState *rng){
    int i, tid;
    double point_in_circle = 0., x = 0., y = 0.;
    //thread id global, convention par ligne / grille 2D
    tid = blockIdx.x * blockDim.x + threadIdx.x;
    hiprand_init(clock64(), tid, 0, &rng[tid]);

    for (i=0; i<N; i++){
        x = hiprand_uniform(&rng[tid]);
        y = hiprand_uniform(&rng[tid]);
        point_in_circle += (double)(x*x +y*y <=1);
    }
    // 1résultat / thread
    d_in_circle[tid] = (double) 4.0*point_in_circle / N;
}

int main(){
    srand(1);
    double in_circle [NB_BLOCK * BLOCK_SIZE] = {0.};
    double *d_in_circle;
    double pi = 0;

    // alloue une copie des curand cote device
    hiprandState *d_rng;
    hipMalloc( (void **)&d_rng, NB_BLOCK * BLOCK_SIZE * sizeof(hiprandState));
    // alloue une copie des result cote device (gpu)
    hipMalloc((void **)&d_in_circle, NB_BLOCK * BLOCK_SIZE* sizeof(double));
    hipMemcpy(d_in_circle, in_circle, NB_BLOCK * BLOCK_SIZE * sizeof(double), hipMemcpyHostToDevice);
    clock_t start = clock();

    //1D ( 1 argument -> automatic Idx.x), 1 block , 1 thread
    //<<<NbBlocks,NBThreads/Block>>>
    cuda_inCircle<<<NB_BLOCK,BLOCK_SIZE>>>(d_in_circle, d_rng);
    hipDeviceSynchronize();
 
    hipMemcpy(in_circle, d_in_circle,  NB_BLOCK * BLOCK_SIZE * sizeof(double), hipMemcpyDeviceToHost);
    // "reduction"  = somme des resultats
    for (int i = 0; i < NB_BLOCK * BLOCK_SIZE; i++)
    {
       pi += in_circle[i];
    }
    pi /= NB_BLOCK * BLOCK_SIZE;
    clock_t end = clock();
    double total = (double) (end - start) / CLOCKS_PER_SEC;
    printf("pi = %.12f , calculated in %.3f seconds with %d block and %d threads for %d iterations/thread\n", pi, total, NB_BLOCK, BLOCK_SIZE, N*NB_BLOCK*BLOCK_SIZE);
    hipFree(d_in_circle);
    return 0;
}