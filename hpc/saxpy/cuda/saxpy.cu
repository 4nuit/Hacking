#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <time.h>

#define BLOCK_SIZE 32
#define NB_BLOCK 16
#define n 10 // Matrix size (10x10 in this example)
#define N (n*n)

__global__ void saxpy(double *c, double *a, double *b) {
    // Grid formula
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // Kernel only support 1D - reduction for each C coeff
    if (row < n && col < n) {
        double sum = 0.0;
        for (int k = 0; k < n; k++) {
            sum += a[row * n + k] * b[k * n + col];
        }
        c[row * n + col] = sum;
    }
}

int main() {
    double *d_c, *d_a, *d_b;
    // Set matrices as 1D blocks of threads (tab contiguous indexing)
    double *c = (double *)calloc(n * n, sizeof(double));
    double *a = (double *)malloc(n * n * sizeof(double));
    double *b = (double *)malloc(n * n * sizeof(double));

    // Initialize matrices a and b
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            a[i * n + j] = 1.0;  // Flattened indexing
            b[i * n + j] = 2.0;
        }
    }

    // Allocate memory on the GPU
    hipMalloc((void **)&d_a, n * n * sizeof(double));
    hipMalloc((void **)&d_b, n * n * sizeof(double));
    hipMalloc((void **)&d_c, n * n * sizeof(double));

    // Copy data from host to device
    hipMemcpy(d_a, a, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, n * n * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_c, c, n * n * sizeof(double), hipMemcpyHostToDevice);

    // Define grid and block sizes
    dim3 threadsPerBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 numBlocks((n + BLOCK_SIZE - 1) / BLOCK_SIZE, (n + BLOCK_SIZE - 1) / BLOCK_SIZE);

    // Print configuration
    printf("numBlocks: {%d, %d, %d}, threadsPerBlock: {%d, %d, %d}\n", numBlocks.x, numBlocks.y, numBlocks.z, threadsPerBlock.x, threadsPerBlock.y, threadsPerBlock.z);

    // Measure the computation time
    clock_t start = clock();

    // Launch kernel
    saxpy<<<numBlocks, threadsPerBlock>>>(d_c, d_a, d_b);
    hipDeviceSynchronize();

    // Copy result back to host
    hipMemcpy(c, d_c, n * n * sizeof(double), hipMemcpyDeviceToHost);

    clock_t end = clock();
    double total = (double)(end - start) / CLOCKS_PER_SEC;

    // Print result matrix
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            printf("%.2f ", c[i * n + j]);
        }
        printf("\n");
        if (i >= 10) break;
    }

    printf("Time: %.3f seconds\n", total);

    // Free memory
    free(c);
    free(a);
    free(b);
    hipFree(d_c);
    hipFree(d_a);
    hipFree(d_b);

    return 0;
}
